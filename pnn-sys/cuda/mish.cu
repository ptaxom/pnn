#include "hip/hip_runtime.h"
#include "kernels.h"

// #TODO: Add more accurate estimation
dim3 get_gridsize(size_t elements){
    unsigned int required_blocks = (elements + BLOCK_SIZE - 1) / BLOCK_SIZE;
    if(required_blocks <= 65535){
        return {required_blocks, 1, 1};
    }
    unsigned int proposed_width = ceil(sqrt(required_blocks));
    unsigned int required_height = (required_blocks - proposed_width + 1) / proposed_width;
    return {proposed_width, required_height, 1};
}

template<typename T>
__device__ T mish(T x) {
    T e = exp(x);
    T n = e * e + 2 * e;
    if (x <= -0.6f)
        return x * n / (n + 2);
    return x - 2 * x / (n + 2);
}

template<>
__device__ __half mish(__half x) {
    __half e = hexp(x);
    half HALF_TWO = __float2half(2.f);
    half n = __hadd(__hmul(e, e), __hmul(e, HALF_TWO));
    half n2 = __hadd(n, HALF_TWO);
    if (__hle(x, -0.6f))
        return __hmul(x , __hdiv(n, n2));
    return __hsub(x, __hmul(2.f , __hdiv(x, n2)));
}

template<>
__device__ float mish(float x) {
    float e = __expf(x);
    float n = e * e + 2 * e;
    if (x <= -0.6f)
        return x * __fdividef(n, n + 2);
    return x - 2 * __fdividef(x, n + 2);
}



template<typename T>
__global__ void activation_mish_kernel(T* data, int elements) {
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i < elements) {
        data[i] = mish<T>(data[i]);
    }
}

template<typename T>
hipError_t activation_mish(void* data, size_t elements, hipStream_t stream) {
    activation_mish_kernel<T><<<get_gridsize(elements), BLOCK_SIZE, 0, stream >>>(
        static_cast<T*>(data), 
        static_cast<int>(elements)
    );
    return hipGetLastError();
}

hipError_t activation_mish_fp16(void* data, size_t elements, hipStream_t stream){
    return activation_mish<__half>(data, elements, stream);
}

hipError_t activation_mish_fp32(void* data, size_t elements, hipStream_t stream){
    return activation_mish<float>(data, elements, stream);
}

hipError_t activation_mish_fp64(void* data, size_t elements, hipStream_t stream) {
    return activation_mish<double>(data, elements, stream);
}