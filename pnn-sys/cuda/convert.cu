#include "hip/hip_runtime.h"
#include "kernels.h"
#include <hipDNN.h>

// template magic not work here :(
__global__ void convert_kernel_half2float(float* output, half* input, int n_elements) {
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i < n_elements) {
        output[i] = __half2float(input[i]);
    }
}

__global__ void convert_kernel_float2half(half* output, float* input, int n_elements) {
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i < n_elements) {
        output[i] = __half2float(input[i]);
    }
}

hipError_t cvt_ptr_data(void* output, void* input, size_t n_elements, size_t otype, size_t itype, hipStream_t stream) {
    if (otype == itype)
        return hipSuccess;

    if (otype == HIPDNN_DATA_FLOAT && itype == HIPDNN_DATA_HALF) {
        convert_kernel_half2float<<<get_gridsize(n_elements), BLOCK_SIZE, 0, stream >>>(
            static_cast<float*>(output), 
            static_cast<half*>(input), 
            static_cast<int>(n_elements)
        );
        return hipGetLastError();
    }

    if (otype == HIPDNN_DATA_HALF && itype == HIPDNN_DATA_FLOAT){
        convert_kernel_float2half<<<get_gridsize(n_elements), BLOCK_SIZE, 0, stream >>>(
            static_cast<half*>(output), 
            static_cast<float*>(input), 
            static_cast<int>(n_elements)
        );
        return hipGetLastError();
    }
    
    return hipErrorInvalidValue;
    
}